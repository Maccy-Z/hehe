#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2011 - 2024 NVIDIA CORPORATION. All Rights Reserved.
//
// SPDX-License-Identifier: BSD-3-Clause

#include <amgx_cublas.h>
#include <blas.h>
#include <chrono>
#include <cutil.h>
#include <norm.h>
#include <util.h>
#include <cusp/blas.h>
#include <solvers/fgmres_solver.h>
#include "solvers/fgmres_utils.h"

//TODO remove synchronization from this module by moving host operations to the device


namespace amgx
{

//init the frist vector
template <class TConfig>
void KrylovSubspaceBuffer<TConfig>::setup(int N_dim, int restart_iters)
{
    new_basis = new thrust::device_vector<float>(N_dim);
    V_matrix = new CudaMatrix(N_dim, restart_iters);
    H_Matrix = new CudaMatrix(restart_iters+1, restart_iters+1);

    this->N_dim = N_dim;
    this->restart_iters = restart_iters;
    this->iteration = -1;
}


template< class T_Config>
FGMRES_Solver<T_Config>::FGMRES_Solver( AMG_Config &cfg, const std::string &cfg_scope ) :
    Solver<T_Config>(cfg, cfg_scope), m_preconditioner(0)

{
    std::string solverName, new_scope, tmp_scope;
    cfg.getParameter<std::string>("preconditioner", solverName, cfg_scope, new_scope);

    if (solverName.compare("NOSOLVER") == 0)
    {
        use_preconditioner = false;
        m_preconditioner = NULL;
    }
    else
    {
        use_preconditioner = true;
        m_preconditioner = SolverFactory<T_Config>::allocate( cfg, cfg_scope, "preconditioner" );
    }

    m_restart = cfg.AMG_Config::template getParameter<int>("gmres_n_restart", cfg_scope);

    e_vect = new thrust::device_vector<float>(m_restart+1);

    // Init least squares solver
    hipblasHandle_t handle_cublas = Cublas::get_handle();
    hipsolverHandle_t hanlde_cusolver = nullptr;
    hipsolverDnCreate(&hanlde_cusolver);
    lstsq_solver = new LeastSquaresSolver(hanlde_cusolver, handle_cublas, m_restart+1, m_restart);
}

template<class T_Config>
FGMRES_Solver<T_Config>::~FGMRES_Solver()
{
    if (use_preconditioner) { delete m_preconditioner; }
    delete lstsq_solver;
    delete e_vect;
}

template<class T_Config>
void
FGMRES_Solver<T_Config>::printSolverParameters() const
{
    std::cout << "gmres_n_restart=" << this->m_restart << std::endl;

    if (use_preconditioner)
    {
        std::cout << "preconditioner: " << this->m_preconditioner->getName() << " with scope name: " << this->m_preconditioner->getScope() << std::endl;
    }
}

template<class T_Config>
void
FGMRES_Solver<T_Config>::solver_setup(bool reuse_matrix_structure)
{
    if (use_preconditioner)
    {
        m_preconditioner->setup( *this->m_A, reuse_matrix_structure );
    }

    ViewType oldView = this->m_A->currentView();
    this->m_A->setViewExterior();
    //should we warn the user about the extra computational work?
    // printf("m_nrm.size() = %d, m_use_scalar_norm = %d, m_norm_type = %d\n", this->m_nrm.size(), this->m_use_scalar_norm, this->m_norm_type);
    use_scalar_L2_norm = (this->m_nrm.size() == 1 || this->m_use_scalar_norm) && this->m_norm_type == L2;
    m_dim = this->m_A->get_num_cols();
    subspace.setup(this->m_A->get_num_cols(), this->m_restart);

    this->m_A->setView(oldView);
}

template<class T_Config>
void
FGMRES_Solver<T_Config>::solve_init( VVector &b, VVector &x, bool xIsZero )
{
    //init residual, even if we don't plan to use it, we might need it, so make sure we have enough memory to store it now
    // residual.resize( b.size() );
    // residual.set_block_dimx( 1 );
    // residual.set_block_dimy( this->m_A->get_block_dimy() );
    // residual.dirtybit = 1;
    // residual.delayed_send = 1;
}




//Run preconditioned GMRES
template<class T_Config>
AMGX_STATUS
FGMRES_Solver<T_Config>::solve_iteration( VVector &b, VVector &x, bool xIsZero )
{
    /*using Clock = std::chrono::steady_clock;
    using TimePoint = std::chrono::time_point<Clock>;
    using Duration = std::chrono::duration<double, std::milli>; // milliseconds
    TimePoint start, end;
    Duration duration;


    hipDeviceSynchronize();
    start = Clock::now();
    hipDeviceSynchronize();
    end = Clock::now();
    duration = end - start;
    std::cout << "main time: " << duration.count() << " ms\n";*/

    // AMGX_STATUS conv_stat = AMGX_ST_CONVERGED;
    hipblasHandle_t cublas_handle = Cublas::get_handle();

    int m = this->m_curr_iter % m_restart;  //Iterations between restarts, do we still need restart?

    auto& new_basis = *subspace.new_basis;
    float* new_basis_ptr = thrust::raw_pointer_cast(new_basis.data());
    auto& V = *subspace.V_matrix;
    auto& H = *subspace.H_Matrix;
    float* e_vect_ptr = thrust::raw_pointer_cast(e_vect->data());
    auto* x_ptr = (float*)thrust::raw_pointer_cast(x.data());

    // A matrix
    auto& A = dynamic_cast<Matrix<T_Config>&>(*this->m_A);

    if (m == 0){
        //initialize gmres
        // A never ever changes, but set once per iteration anyway.
        sp_axpy.set_matrix(A);

        subspace.iteration = 0;
        // compute initial residual r0 = b - Ax
        thrust::copy(b.begin(), b.end(), new_basis.begin());
        sp_axpy.axpy(x_ptr, new_basis_ptr, -1.0f, 1.0f);

        // normalize initial residual
        float* beta = compute_L2_norm(new_basis);
        scale_vector(new_basis, beta);
        V.setColumn(m, new_basis_ptr);

        // e = [beta, 0, 0, ...]
        thrust::fill(e_vect->begin(), e_vect->end(), 0);
        hipMemcpy(e_vect_ptr, beta, sizeof(float), hipMemcpyDeviceToDevice);

    }

    // Copy new_basis into V
    subspace.iteration = m;

    // Run one iteration of preconditioner with zero initial guess and v_m as rhs, i.e. solve Az_m=v_m
    // copy(subspace.V(m), subspace.Z(m), offset, size);

    //obtain v_m+1 := A*z_m
    sp_axpy.axpy(V.getColPtr(m), new_basis_ptr, 1.0f, 0.0f);


    // Compute next vector in the basis using Gram Schmidt and entry in Hessenberg matrix
    gram_schmidt_step(V.getColPtr(0), m_dim, m, H.getDevicePointer(), m_restart+1, new_basis_ptr);


    //H(m+1,m) = || v_m+1 ||
    float* norm = compute_L2_norm(new_basis);
    H.set_element_device(m+1, m, norm);
    // //normalize v_m+1
    scale_vector(new_basis, norm);

    if (m < m_restart-1)
    {
        V.setColumn(m+1, new_basis_ptr);
    }



    // // If reached restart limit or last iteration or if converged, compute x vector
    //    if ( !update_x_every_iteration && (m == m_R - 1 || this->is_last_iter() || isDone(conv_stat) ))
    if (this->is_last_iter() || m == m_restart - 1 )
    {

        lstsq_solver->lstsq_solve(H.getDevicePointer(), e_vect_ptr);


        const float one = 1.0f;
        // x = x + A e_vect
        hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST);
        hipblasSgemv(cublas_handle,
                    HIPBLAS_OP_N, // No transpose
                    V.getRows(),           // Number of rows of A
                    V.getCols(),           // Number of columns of A
                    &one,      // alpha
                    V.getDevicePointer(),         // A
                    V.getRows(),           // leading dimension of A
                    e_vect_ptr,         // y
                    1,           // stride of y
                    &one,       // beta
                    x_ptr,         // x
                    1            // stride of x
        );


        // printvec(x_ptr, 10);


    }
    //
    //
    // A.setView(oldView);
    //
    //
    // std::exit(69);
    hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST);
    return AMGX_ST_NOT_CONVERGED;
    //return Base::m_monitor_convergence ? conv_stat : AMGX_ST_CONVERGED;
}

template<class T_Config>
void
FGMRES_Solver<T_Config>::solve_finalize( VVector &b, VVector &x )
{
    // residual.resize(0);
}

/****************************************
* Explict instantiations
***************************************/
#define AMGX_CASE_LINE(CASE) template class FGMRES_Solver<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

} // namespace amgx
